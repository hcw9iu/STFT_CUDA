#include "hip/hip_runtime.h"
// stft_cufft_hanning.cu
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <math.h>

namespace py = pybind11;

__global__ void process_segments(float* input, hipFloatComplex* output,
                               float* hanning_window, 
                               int signal_length, int window_size, 
                               int hop_size, int fft_size,
                               int num_segments) {
    int m = blockIdx.x;
    int tid = threadIdx.x;
    
    if (m < num_segments && tid < window_size) {
        int input_idx = m * hop_size + tid;
        if (input_idx < signal_length) {
            float sample = input[input_idx] * hanning_window[tid];
            output[m * fft_size + tid].x = sample;
            output[m * fft_size + tid].y = 0.0f;
        }
    }
}

__global__ void normalize_fft(hipfftComplex* data, int size, float Fs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float norm = Fs;
        data[idx].x /= norm;
        data[idx].y /= norm;
    }
}

__global__ void transpose(hipFloatComplex* d_output, hipFloatComplex* temp, int num_segments, int fft_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < num_segments && j < (fft_size/2 + 1)) {
        temp[j * num_segments + i] = d_output[i * (fft_size/2 + 1) + j];
    }
}

void performTranspose(hipFloatComplex* d_output, hipFloatComplex* temp, int num_segments, int fft_size) {
    dim3 blockSize(16, 16);
    dim3 gridSize((num_segments + blockSize.x - 1) / blockSize.x, (fft_size/2 + 1 + blockSize.y - 1) / blockSize.y);

    transpose<<<gridSize, blockSize>>>(d_output, temp, num_segments, fft_size);
    hipDeviceSynchronize();
}

void stft_cufft_hanning(float* input, hipFloatComplex* output, int signal_length, int window_size, 
int hop_size, int fft_size, int Fs) {
    int num_segments = 1 + (signal_length - window_size) / window_size;
    
    // 分配設備內存
    float *d_input, *d_hanning_window;
    hipFloatComplex *d_output;
    
    hipMalloc(&d_input, signal_length * sizeof(float));
    hipMalloc(&d_output, num_segments * fft_size * sizeof(hipFloatComplex));
    hipMalloc(&d_hanning_window, window_size * sizeof(float));
    
    // [!] 創建並初始化 hanning window
    float* h_hanning = (float*)malloc(window_size * sizeof(float));
    for(int i = 0; i < window_size; i++) {
        h_hanning[i] = 0.5f * (1.0f - cosf(2.0f * M_PI * i / (window_size - 1)));
    }
    hipMemcpy(d_hanning_window, h_hanning, window_size * sizeof(float), 
               hipMemcpyHostToDevice);
    free(h_hanning);
    
    // 複製輸入數據
    hipMemcpy(d_input, input, signal_length * sizeof(float), 
               hipMemcpyHostToDevice);
    
    // 設置 grid 和 block 維度
    dim3 block(256);
    dim3 grid((num_segments + block.x - 1) / block.x);
    
    // 啟動 kernel
    process_segments<<<grid, block>>>(d_input, d_output, d_hanning_window,
                                    signal_length, window_size, hop_size,
                                    fft_size, num_segments);
    
    // 設置並執行 FFT
    hipfftHandle plan;
    hipfftResult result = hipfftPlan1d(&plan, fft_size, HIPFFT_R2C, num_segments);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan creation failed\n");
    }

    result = hipfftExecR2C(plan, (hipfftReal*)d_input, (hipfftComplex*)d_output);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT error: ExecR2C failed\n");
    }
    
    // [!] FFT 後正規化
    int total_elements = num_segments * (fft_size/2 + 1);
    dim3 normalize_block(256);
    dim3 normalize_grid((total_elements + normalize_block.x - 1) / normalize_block.x);
    normalize_fft<<<normalize_grid, normalize_block>>>((hipfftComplex*)d_output, total_elements, (float)Fs);
    
    // [!] 確保正規化完成
    hipDeviceSynchronize();
    
    //size_t src_pitch = num_segments * sizeof(hipFloatComplex);
    //size_t dst_pitch = (fft_size/2 + 1) * sizeof(hipFloatComplex);
    //size_t width = sizeof(hipFloatComplex);  // 每次複製一個複數值
    //size_t height = num_segments;  // 行數，即時間段的數量

    //hipMemcpy2D(output, dst_pitch, d_output, src_pitch, width, height, hipMemcpyDeviceToHost);


    hipFloatComplex* temp;
    hipMalloc(&temp, num_segments * (fft_size/2+1) * sizeof(hipFloatComplex));

    performTranspose(d_output, temp, num_segments, fft_size);

    hipMemcpy(output, temp, num_segments * (fft_size/2+1) * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    

    // [!] 釋放所有內存
    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_hanning_window);
    
    // [!] 確保所有 CUDA 操作完成
    hipDeviceSynchronize();

}

// pybind11 包裝函數
void stft_pybind(py::array_t<float> inputArray, py::array_t<std::complex<float>> outputArray, int window_size, int hop_size, int fft_size, int Fs) {
    auto inputBuf = inputArray.request();
    auto outputBuf = outputArray.request();

    float* input = static_cast<float*>(inputBuf.ptr);
    //hipFloatComplex* output = static_cast<hipFloatComplex*>(outputBuf.ptr);
    std::complex<float>* output = static_cast<std::complex<float>*>(outputBuf.ptr);

    int signal_length = inputBuf.shape[0];

    stft_cufft_hanning(input, reinterpret_cast<hipfftComplex*>(output), signal_length, window_size, hop_size, fft_size, Fs);
}

PYBIND11_MODULE(stft_cuda, m) {
    m.def("stft", &stft_pybind, "Short Time Fourier Transform using cuFFT with Hanning window");
}
