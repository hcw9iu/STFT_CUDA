#include "hip/hip_runtime.h"
// stft_cufft_hanning.cu
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <math.h>

namespace py = pybind11;

__global__ void process_segments(float* input, float* output,
                               float* hanning_w, 
                               int sig_len, int win_size, 
                               int hop_size,
                               int segs) {
    int m = blockIdx.x;
    int tid = threadIdx.x;
    
    if (m < segs && tid < win_size) {
        int input_idx = m * hop_size + tid;
        if (input_idx < sig_len) {
            float sample = input[input_idx] * hanning_w[tid];
            output[m * win_size + tid] = sample;
            //output[m * fft_size + tid].y = 0.0f;
        }
    }
}

__global__ void normalize_fft(hipfftComplex* data, int size, float Fs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float norm = Fs;
        data[idx].x /= norm;
        data[idx].y /= norm;
    }
}

__global__ void transpose(hipFloatComplex* d_output, hipFloatComplex* d_output_T, int segs, int fft_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < segs && j < (fft_size/2 + 1)) {
        d_output_T[j * segs + i] = d_output[i * (fft_size/2 + 1) + j];
    }
}

void performTranspose(hipFloatComplex* d_segs, hipFloatComplex* d_output_T, int segs, int fft_size) {
    dim3 blockSize(16, 16);
    dim3 gridSize((segs + blockSize.x - 1) / blockSize.x, (fft_size/2 + 1 + blockSize.y - 1) / blockSize.y);

    transpose<<<gridSize, blockSize>>>(d_segs, d_output_T, segs, fft_size);
    hipDeviceSynchronize();
}

void stft_cufft_hanning(float* input, hipFloatComplex* output, int sig_len, int win_size, 
int hop_size, int fft_size, int Fs) {
    int segs = 1 + (sig_len - win_size) / hop_size;
    
    // 分配設備內存
    //float *d_input, *d_hanning_win, *d_segs;
    float *d_input, *d_hanning_win, *d_output_B;
    hipfftComplex *d_output, *d_output_T;
    
    //size_t fft_buffer_size = segs * fft_size * sizeof(float);  // FFT 緩衝區大小
    size_t output_samples = segs * (fft_size/2+1) * sizeof(hipfftComplex);  // 輸出緩衝區大小
    size_t samples = segs * fft_size * sizeof(hipfftComplex); 
    size_t samples_B = segs * win_size * sizeof(float); 

    // 分配足夠的空間用於 FFT
    hipMalloc(&d_input, sig_len * sizeof(float));  // 修改：分配 FFT 緩衝區大小
    hipMalloc(&d_output_B, samples_B);  // 修改：分配 FFT 緩衝區大小
    hipMalloc(&d_output, samples);  // FFT 輸出需要相同大小
    hipMalloc(&d_output_T, output_samples);  // 最終輸出緩衝區
    hipMalloc(&d_hanning_win, win_size * sizeof(float));

    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        printf("CUDA error before FFT plan: %s\n", hipGetErrorString(cudaErr));
        return;
    }
    
    // set init value = 0
    hipMemset(d_input, 0, sig_len * sizeof(float));
    hipMemset(d_output, 0, samples);
    hipMemset(d_output_B, 0, samples_B);
    //hipMemset(d_segs, 0, fft_buffer_size);
    
    // create and init hanning window
    float* h_hanning = (float*)malloc(win_size * sizeof(float));
    for(int i = 0; i < win_size; i++) {
        h_hanning[i] = 0.5f * (1.0f - cosf(2.0f * M_PI * i / (win_size - 1)));
    }
    hipMemcpy(d_hanning_win, h_hanning, win_size * sizeof(float), 
               hipMemcpyHostToDevice);
    free(h_hanning);
    
    // copy input data
    hipMemcpy(d_input, input, sig_len * sizeof(float), hipMemcpyHostToDevice);
    
    // set grid and block dimension
    dim3 block(win_size);
    dim3 grid(segs);
    
    // launch kernel
    process_segments<<<grid, block>>>(d_input, d_output_B, d_hanning_win,
                                    sig_len, win_size, hop_size, segs);
    
    // set and execute FFT
    hipfftHandle plan;
    hipfftResult result = hipfftPlan1d(&plan, fft_size, HIPFFT_R2C, segs);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan creation failed\n");
    }

    result = hipfftExecR2C(plan, d_output_B, d_output);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT error: ExecR2C failed\n");
    }
    
    // FFT normalization
    int total_elements = segs * (fft_size/2 + 1);
    dim3 normalize_block(256);
    dim3 normalize_grid((total_elements + normalize_block.x - 1) / normalize_block.x);
    normalize_fft<<<normalize_grid, normalize_block>>>(d_output, total_elements, (float)Fs);
    
    // ensure normalization completed
    hipDeviceSynchronize();
    
    //size_t src_pitch = num_segments * sizeof(hipFloatComplex);
    //size_t dst_pitch = (fft_size/2 + 1) * sizeof(hipFloatComplex);
    //size_t width = sizeof(hipFloatComplex);  // 每次複製一個複數值
    //size_t height = num_segments;  // 行數，即時間段的數量

    //hipMemcpy2D(output, dst_pitch, d_output, src_pitch, width, height, hipMemcpyDeviceToHost);


    //hipFloatComplex* temp;
    //hipMalloc(&temp, segs * (fft_size/2+1) * sizeof(hipFloatComplex));

    performTranspose(d_output, d_output_T, segs, fft_size);

    hipMemcpy(output, d_output_T, segs * (fft_size/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    

    // 釋放所有內存
    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_output_B);
    hipFree(d_hanning_win);
    hipFree(d_output_T);
    
    // 確保所有 CUDA 操作完成
    hipDeviceSynchronize();

}

// pybind11 wrapper
void stft_pybind(py::array_t<float> inputArray, py::array_t<std::complex<float>> outputArray, int win_size, int hop_size, int fft_size, int Fs) {
    auto inputBuf = inputArray.request();
    auto outputBuf = outputArray.request();

    float* input = static_cast<float*>(inputBuf.ptr);
    //hipFloatComplex* output = static_cast<hipFloatComplex*>(outputBuf.ptr);
    std::complex<float>* output = static_cast<std::complex<float>*>(outputBuf.ptr);

    int sig_len = inputBuf.shape[0];

    stft_cufft_hanning(input, reinterpret_cast<hipfftComplex*>(output), sig_len, win_size, hop_size, fft_size, Fs);
}

PYBIND11_MODULE(stft_cuda, m) {
    m.def("stft", &stft_pybind, "Short Time Fourier Transform using cuFFT with Hanning window");
}