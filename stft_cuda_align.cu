#include "hip/hip_runtime.h"
// stft_cufft_hanning.cu
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <math.h>

#define SHARED_SIZE 1024

namespace py = pybind11;

__global__ void process_segments(float* input, float* output,
                               float* hanning_w, 
                               int sig_len, int win_size, 
                               int hop_size,
                               int segs) {
    __shared__ float hanning_w_shared[SHARED_SIZE];
    int m = blockIdx.x;
    int tid = threadIdx.x;

    if (tid < win_size) {
        hanning_w_shared[tid] = hanning_w[tid];
    }
    __syncthreads();
    
    if (m < segs && tid < win_size) {
        int input_idx = m * hop_size + tid;
        if (input_idx < sig_len) {
            float sample = __ldg(&input[input_idx]) * hanning_w_shared[tid];
            output[m * win_size + tid] = sample;
        }
    }
}

__global__ void normalize_fft(hipfftComplex* data, int size, float Fs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float norm = Fs;
        data[idx].x /= norm;
        data[idx].y /= norm;
    }
}

__global__ void transpose(hipFloatComplex* d_output, hipFloatComplex* d_output_T, int segs, int fft_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < segs && j < (fft_size/2 + 1)) {
        d_output_T[j * segs + i] = d_output[i * (fft_size/2 + 1) + j];
    }
}

void performTranspose(hipFloatComplex* d_segs, hipFloatComplex* d_output_T, int segs, int fft_size) {
    dim3 blockSize(16, 16);
    dim3 gridSize((segs + blockSize.x - 1) / blockSize.x, (fft_size/2 + 1 + blockSize.y - 1) / blockSize.y);

    transpose<<<gridSize, blockSize>>>(d_segs, d_output_T, segs, fft_size);
    hipDeviceSynchronize();
}

void stft_cufft_hanning(float* input, hipFloatComplex* output, int sig_len, int win_size, 
int hop_size, int fft_size, int Fs) {
    int segs = 1 + (sig_len - win_size) / hop_size;
    
    // device memory
    float *d_input, *d_hanning_win, *d_output_B;
    hipfftComplex *d_output, *d_output_T;
    
    // set output size
    size_t output_samples = segs * (fft_size/2+1) * sizeof(hipfftComplex);  
    size_t samples = segs * fft_size * sizeof(hipfftComplex); 
    size_t samples_B = segs * win_size * sizeof(float); 

    // allocate device memory
    hipMalloc(&d_input, sig_len * sizeof(float));  
    hipMalloc(&d_output_B, samples_B);  // input segment handler buffer
    hipMalloc(&d_output, samples);  // FFT output buffer
    hipMalloc(&d_output_T, output_samples);  // transposed final output buffer
    hipMalloc(&d_hanning_win, win_size * sizeof(float));

    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        printf("CUDA error before FFT plan: %s\n", hipGetErrorString(cudaErr));
        return;
    }
    
    // set init value = 0
    hipMemset(d_input, 0, sig_len * sizeof(float));
    hipMemset(d_output, 0, samples);
    hipMemset(d_output_B, 0, samples_B);
    //hipMemset(d_segs, 0, fft_buffer_size);
    
    // create and init hanning window
    float* h_hanning = (float*)malloc(win_size * sizeof(float));
    for(int i = 0; i < win_size; i++) {
        h_hanning[i] = 0.5f * (1.0f - cosf(2.0f * M_PI * i / (win_size - 1)));
    }
    hipMemcpy(d_hanning_win, h_hanning, win_size * sizeof(float), 
               hipMemcpyHostToDevice);
    free(h_hanning);
    
    // copy input data
    hipMemcpy(d_input, input, sig_len * sizeof(float), hipMemcpyHostToDevice);
    
    // set grid and block dimension
    dim3 block(win_size);
    dim3 grid(segs);
    
    // launch kernel
    process_segments<<<grid, block>>>(d_input, d_output_B, d_hanning_win,
                                    sig_len, win_size, hop_size, segs);
    
    // set and execute FFT
    hipfftHandle plan;
    hipfftResult result = hipfftPlan1d(&plan, fft_size, HIPFFT_R2C, segs);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan creation failed\n");
    }

    result = hipfftExecR2C(plan, d_output_B, d_output);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT error: ExecR2C failed\n");
    }
    
    // FFT normalization
    int total_elements = segs * (fft_size/2 + 1);
    dim3 normalize_block(256);
    dim3 normalize_grid((total_elements + normalize_block.x - 1) / normalize_block.x);
    normalize_fft<<<normalize_grid, normalize_block>>>(d_output, total_elements, (float)Fs);
    
    // ensure normalization completed
    hipDeviceSynchronize();

    performTranspose(d_output, d_output_T, segs, fft_size);

    hipMemcpy(output, d_output_T, segs * (fft_size/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    

    // release memory
    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_output_B);
    hipFree(d_hanning_win);
    hipFree(d_output_T);
    
    // ensure all CUDA operations completed
    hipDeviceSynchronize();

}

// pybind11 wrapper
void stft_pybind(py::array_t<float> inputArray, py::array_t<std::complex<float>> outputArray, int win_size, int hop_size, int fft_size, int Fs) {
    auto inputBuf = inputArray.request();
    auto outputBuf = outputArray.request();

    float* input = static_cast<float*>(inputBuf.ptr);
    std::complex<float>* output = static_cast<std::complex<float>*>(outputBuf.ptr);

    int sig_len = inputBuf.shape[0];

    stft_cufft_hanning(input, reinterpret_cast<hipfftComplex*>(output), sig_len, win_size, hop_size, fft_size, Fs);
}

PYBIND11_MODULE(stft_cuda_align, m) {
    m.def("stft", &stft_pybind, "Short Time Fourier Transform using cuFFT with Hanning window");
}